#include "hip/hip_runtime.h"
#include "helpers.h"
#include "ping.h"

#include <math.h>
#include <cmath>

__global__ void kernel() {
  std::sin(0.); // nvcc - ok
  std::sin(0);  // nvcc - error, because no std::sin(int) override is available.
  sin(0);       // nvcc - same as above.

  sinf(0.);       // nvcc - ok
  std::sinf(0.);  // nvcc - no such function
}

void lawrencium::cuda::launch() {
  kernel<<<1, 1>>>();
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}