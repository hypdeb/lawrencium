#include "hip/hip_runtime.h"
#include "helpers.h"
#include "ping.h"

#include <math.h>
#include <cmath>
#include <hip/hip_runtime.h>

__global__ void kernel() {
  sin(0.); // nvcc - ok
}

void lawrencium::cuda::launch() {
  kernel<<<1, 1>>>();
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}