#include "hip/hip_runtime.h"
#include "helpers.h"
#include "ping.h"

#include <math.h>
#include <cmath>

__global__ void kernel() {
  std::sin(0.); // nvcc - ok
}

void lawrencium::cuda::launch() {
  kernel<<<1, 1>>>();
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}