#include "hip/hip_runtime.h"
#include "matrix_multiply.h"

#include "hip/hip_runtime.h"
#include ""

/**
 * \brief The thread block size.
 */
constexpr int block_size = 1;

__global__ void MatMulKernel(lawrencium::ccpg::library::matrix left_hand,
                             lawrencium::ccpg::library::matrix right_hand,
                             lawrencium::ccpg::library::matrix result) {
  float result_value = 0;
  const auto row = blockIdx.y * blockDim.y + threadIdx.y;
  const auto col = blockIdx.x * blockDim.x + threadIdx.x;
  for (auto e = 0; e < left_hand.width; ++e) {
    result_value += left_hand.elements[row * left_hand.width + e] *
                    right_hand.elements[e * right_hand.width + col];
  }

  result.elements[row * result.width + col] = result_value;
}

void lawrencium::ccpg::library::MatMul(const lawrencium::ccpg::library::matrix lh, const lawrencium::ccpg::library::matrix rh, lawrencium::ccpg::library::matrix result) {
  // TODO: extract this creation of device matrix.
  // TODO: understand why it's okay for the height and width to be host-side.
  lawrencium::ccpg::library::matrix device_lh = {
      .width = lh.width,
      .height = lh.height,
  };
  const auto lh_elements_memory_size = static_cast<size_t>(lh.width) *
                                       static_cast<size_t>(lh.height) *
                                       sizeof(float);
  hipMalloc(&device_lh.elements, lh_elements_memory_size);
  hipMemcpy(device_lh.elements, lh.elements, lh_elements_memory_size,
             hipMemcpyHostToDevice);

  lawrencium::ccpg::library::matrix device_rh = {
      .width = rh.width,
      .height = rh.height,
  };
  const auto rh_elements_memory_size = static_cast<size_t>(rh.width) *
                                       static_cast<size_t>(rh.height) *
                                       sizeof(float);
  hipMalloc(&device_rh.elements, rh_elements_memory_size);
  hipMemcpy(device_rh.elements, rh.elements, rh_elements_memory_size,
             hipMemcpyHostToDevice);

  lawrencium::ccpg::library::matrix device_result = {
      .width = result.width,
      .height = result.height,
  };
  const auto result_elements_memory_size = static_cast<size_t>(result.width) *
                                           static_cast<size_t>(result.height) *
                                           sizeof(float);
  hipMalloc(&device_result.elements, result_elements_memory_size);

  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid(rh.width / dimBlock.x, lh.height / dimBlock.y);
  MatMulKernel<<<dimGrid, dimBlock>>>(device_lh, device_rh, device_result);
  hipDeviceSynchronize();
  hipMemcpy(result.elements, device_result.elements,
             result_elements_memory_size, hipMemcpyDeviceToHost);

  hipFree(device_lh.elements);
  hipFree(device_rh.elements);
  hipFree(device_result.elements);
  hipDeviceSynchronize();
}
