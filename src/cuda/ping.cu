#include "hip/hip_runtime.h"
#include "helpers.h"
#include "ping.h"

#include <iostream>

__global__ void kernel() {
  printf("cuda kernel called!\n");
}

void lawrencium::cuda::launch() {
  kernel<<<1, 1>>>();
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}