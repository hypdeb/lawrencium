#include "hip/hip_runtime.h"
#include "basic_kernels.cuh"

__global__ void lawrencium::cuda::add_kernel(float *A, float *B, float *C, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

__global__ void lawrencium::cuda::sum_rows_of_square_matrix(const float *matrix, float* result, const int sideLength) {
  int rowIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if(rowIdx < sideLength) {
    float sum = 0.0;
    for(int colIdx = 0; colIdx < sideLength; colIdx++) {
      sum += matrix[rowIdx * sideLength + colIdx];
    }
    result[rowIdx] = sum;
  }
}

__global__ void lawrencium::cuda::transpose_square_matrix(const float* inMatrix, float* outMatrix, const int sideLength) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idx < sideLength && idy < sideLength) {
    int pos = idy * sideLength + idx;
    int trans_pos = idx * sideLength + idy;
    outMatrix[trans_pos] = inMatrix[pos];
  }
}

__global__ void lawrencium::cuda::all_equal_to(const float* inVector, const int length, const float target, const float tolerance, bool *result) {
  *result = true;
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  const auto upperBound = target + tolerance;
  const auto lowerBound = target - tolerance;
  if (i < length) {

    // I assume that since the operation is idempotent, this is not a problem?
    if (!(inVector[i] <= upperBound && inVector[i] >= lowerBound)) {
      *result = false;
    }
  }
}