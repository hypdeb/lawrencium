#include "helpers.h"
#include "basic_cuda.cuh"
#include "basic_kernels.cuh"

void lawrencium::cuda::cuda_add(float *A, float *B, float *C, int N) {
  const auto size = N * sizeof(float);

  float *d_A;
  CUDA_CHECK(hipMalloc(&d_A, size));
  float *d_B;
  CUDA_CHECK(hipMalloc(&d_B, size));
  float *d_C;
  CUDA_CHECK(hipMalloc(&d_C, size));
  CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  lawrencium::cuda::add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
};

bool lawrencium::cuda::is_doubly_stochastic(const float *matrix, const int sideLength) {
  const auto numElements = sideLength * sideLength;

  // Space required in bytes for a square matrix of size sideLength * sideLength
  const auto matrixSizeInBytes = numElements * sizeof(float);

  // Allocate space for the original matrix.
  float *dMatrix;
  CUDA_CHECK(hipMalloc(&dMatrix, matrixSizeInBytes));

  // Space required in bytes for a square matrix of size sideLength * sideLength
  const auto sumsSizeInBytes = sideLength * sizeof(float);

  // Allocate space for vector of sums of rows / columns.
  float *dSums;
  CUDA_CHECK(hipMalloc(&dSums, sumsSizeInBytes));

  // We start by summing the rows.
  const int threadsPerBlock = 256;
  const int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  lawrencium::cuda::sum_rows_of_square_matrix<<<blocksPerGrid, threadsPerBlock>>>(dMatrix, dSums, sideLength);
  CUDA_CHECK(hipPeekAtLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  // Then we check that all the elements are 1. We need some boolean indicator on GPU, which we then copy back.
  bool *dStochasticity;
  const auto sizeOfBool = sizeof(bool);
  CUDA_CHECK(hipMalloc(&dStochasticity, sizeOfBool));

  bool rowStochasticity;
  lawrencium::cuda::all_equal_to<<<blocksPerGrid, threadsPerBlock>>>(dSums, sideLength, 1.0f, 1e-6f, dStochasticity);
  CUDA_CHECK(hipPeekAtLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemcpy(&rowStochasticity, dStochasticity, sizeOfBool, hipMemcpyDeviceToHost));

  // If it's already not stochastic here, early return. But we need to clean up... #localfunctions :)
  if (!rowStochasticity) {
    CUDA_CHECK(hipFree(dMatrix));
    CUDA_CHECK(hipFree(dSums));
    CUDA_CHECK(hipFree(dStochasticity));
    return false;
  }

  // Allocate space for the transposed matrix, which we need because it is allegedly faster to transpose and sum the rows, than to sum the columns.
  float *dTransposedMatrix;
  CUDA_CHECK(hipMalloc(&dTransposedMatrix, matrixSizeInBytes));

  // We do the transposition.
  lawrencium::cuda::transpose_square_matrix<<<blocksPerGrid, threadsPerBlock>>>(dMatrix, dTransposedMatrix, sideLength);
  CUDA_CHECK(hipPeekAtLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  // Now we can already get rid of the original.
  CUDA_CHECK(hipFree(dMatrix));

  // Then we do the summing, checking equality, and copying back.
  lawrencium::cuda::sum_rows_of_square_matrix<<<blocksPerGrid, threadsPerBlock>>>(dTransposedMatrix, dSums, sideLength);
  CUDA_CHECK(hipPeekAtLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  lawrencium::cuda::all_equal_to<<<blocksPerGrid, threadsPerBlock>>>(dSums, sideLength, 1.0f, 1e-6f, dStochasticity);
  CUDA_CHECK(hipPeekAtLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemcpy(&rowStochasticity, dStochasticity, sizeOfBool, hipMemcpyDeviceToHost));

  // Now we do not have anything more to do on GPU, so we clean up.
  CUDA_CHECK(hipFree(dMatrix));
  CUDA_CHECK(hipFree(dTransposedMatrix));
  CUDA_CHECK(hipFree(dSums));
  CUDA_CHECK(hipFree(dStochasticity));

  // And finally we return the stochasticity of the rows of the transposed matrix / columns of the original matrix.
  return rowStochasticity;
}