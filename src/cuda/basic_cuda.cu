#include "helpers.h"
#include "basic_cuda.cuh"
#include "basic_kernels.cuh"

void lawrencium::cuda::cuda_add(float *A, float *B, float *C, int N) {
  const auto size = N * sizeof(float);

  float *d_A;
  CUDA_CHECK(hipMalloc(&d_A, size));
  float *d_B;
  CUDA_CHECK(hipMalloc(&d_B, size));
  float *d_C;
  CUDA_CHECK(hipMalloc(&d_C, size));
  CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  lawrencium::cuda::add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
};